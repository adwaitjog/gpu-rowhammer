
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test(int* dataD, int* sumD) {
	int x = dataD[0];
	int y = dataD[1];
	int z = dataD[2];
	*sumD = x + y + z;
}

int main() {
	int* dataH = (int*)malloc(sizeof(int)*10);
	for (int i = 0; i < 10; i++) {
		dataH[i] = i; 
	}
	int* dataD;
	hipMalloc((void**)&dataD, sizeof(int)*10);
	hipMemcpy(dataD, dataH, sizeof(int)*10, hipMemcpyHostToDevice);
	int* sumH = (int*)malloc(sizeof(int));
	int* sumD; 
	hipMalloc((void**)&sumD, sizeof(int));
	test<<<1,1>>>(dataD, sumD);
	hipMemcpy(sumH, sumD, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", *sumH);
}
