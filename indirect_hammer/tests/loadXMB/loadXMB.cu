#include "hip/hip_runtime.h"
#include <time.h>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"

// CARD TARGETED : K40c

// 1.5 MB 
const size_t CACHESIZE = 1.5 * (1<<20);

// 32 B
// const size_t CLSIZE = 32;

const size_t intsize = sizeof(int);

void check_error(hipError_t cudaerr) {
    if (cudaerr != hipSuccess) {
        printf("FAILED WITH ERROR: \"%s\".\n", hipGetErrorString(cudaerr));
        exit(-1);
    }   
}

__global__ void fill_cache_stride(int* vals, int size, int stride, int iters) {
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint nthreads = blockDim.x * gridDim.x; 
	int sum;
	for (int t = 0; t < iters; t++) { 
		for (int i = tid*stride; i < size/intsize; i += (nthreads*stride)) {
			int n1 = vals[i];
			//int n2 = vals[thread_i+1];
			sum += n1;
		}
		vals[0] = sum;
		//printf("first kernel\n");
	}
}

__global__ void fill_cache_stride_1thread(int* vals, int size, int stride) {
	//uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	//uint nthreads = blockDim.x * gridDim.x;

	int sum;
	for (int j = 0; j < 100; j++) {
	for (int i = 0; i < size/intsize; i += stride) {
		int n1 = vals[i];
		//int n2 = vals[thread_i+1];
		sum += n1;
	}}
	vals[0] = sum;
	//printf("first kernel\n");

}

int main(int argc, char** argv) {
	if (argc != 6) {
		printf("USAGE: ./loadXMB <# blocks: int> <# threads: int> <size_mult: double (multipler of cache size)> <stride: int> <iterations: int>\n");
	}
	int blocks = atoi(argv[1]);
	int threads = atoi(argv[2]);
	double size_mult = atof(argv[3]);
	int stride = atoi(argv[4]);
	int iters = atoi(argv[5]);
	srand(time(NULL));	

	int size = (int)(size_mult * CACHESIZE);

	int* valsHost = (int*) malloc(size); 
	
	memset(valsHost, 0, size); 
	
	for (int i = 0; i < size/intsize; i++) {
		valsHost[i] = (int)rand();
	} 
	
	int* valsDevice;
 	hipMalloc((void**)&valsDevice, size);
 	hipMemcpy(valsDevice, valsHost, size, hipMemcpyHostToDevice); 
 		
	GpuTimer timer1;
	timer1.Start();
 	fill_cache_stride<<<blocks, threads>>>(valsDevice, size, stride, iters); 
 	//fill_cache_stride_1thread<<<1, 1>>>(valsDevice, size, stride); 
 	check_error(hipDeviceSynchronize());
	timer1.Stop();
 	//check_error(hipDeviceSynchronize());
	printf("blocks: %d | threads: %d  | size_mult: %f | stride: %d | iters: %d\n", blocks, threads, size_mult, stride, iters);
	printf("timer1: %g | val: %d\n", timer1.Elapsed(), valsHost[0]); 
}
